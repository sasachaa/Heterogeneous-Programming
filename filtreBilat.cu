#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Gaussian function
__device__ float gaussian(float x, float sigma) {
    return expf(-(x * x) / (2.0f * sigma * sigma));
}

// fonction appelée depuis le CPU mais exécutée sur le GPU ! 
// ici chaque pixel va être traiter par un thread 
__global__ void bilateral_filter_kernel(
    const unsigned char* d_src,    // image source sur le GPU
    unsigned char* d_dst,          // image destination sur le GPU
    int width, int height, int channels,
    const float* d_spatial_weights, // poids spatiaux sur le GPU
    int d, float sigma_color) 
{   
    // on optimise l'accès à la mémoire globale en lisant les données de manière colonnales lorsqu'on travail sur les colonnes d'une matrice
    // calcul des indices des threads dans une grille 2d de blocs en cuda ces indices permettent de ref les élém d'une matrice stockée en mémoire globale
    // on regarde les pixels voisins dans une petite fenêtre d × d, on calcule les poids et on fait une moyenne pondérée.
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // colonne
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // ligne
    int radius = d / 2;

    // Limiter aux pixels valides (on ne veut pas les bords)
    if (x < radius || x >= width - radius || y < radius || y >= height - radius) return;

    int center_idx = (y * width + x) * channels;

    for (int c = 0; c < channels; c++) {
        float filtered_value = 0.0f;
        float weight_sum = 0.0f;

        float center_val = d_src[center_idx + c];

        // Boucle sur la fenêtre locale
        for (int i = 0; i < d; i++) {
            for (int j = 0; j < d; j++) {
                // Positon des voisins 
                int nx = x + j - radius;
                int ny = y + i - radius;

                if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;
                // on lie la couleur du voisin 
                int neighbor_idx = (ny * width + nx) * channels + c;
                float neighbor_val = d_src[neighbor_idx];
                // on calcule et on fait la moy pondérée
                float spatial = d_spatial_weights[i * d + j];
                float range = gaussian(fabsf(neighbor_val - center_val), sigma_color);
                float weight = spatial * range;

                filtered_value += weight * neighbor_val;
                weight_sum += weight;
            }
        }
        // Et on n'oublie pas de NORMALISER 
        d_dst[center_idx + c] = (unsigned char)(filtered_value / (weight_sum + 1e-6f));
    }
}


// Main function
int main(int argc, char *argv[]) {
    if (argc < 3) {
        // prend le premier argument qui est donc l'image
        printf("Usage: %s <input_image> <output_image>\n", argv[0]);
        return 1;
    }
    // le cpu va charger l'image
    int width, height, channels;
    unsigned char *h_src = stbi_load(argv[1], &width, &height, &channels, 0); // CPU
    if (!h_src) {
        printf("Error loading image!\n");
        return 1;
    }

    if (width <= 5 || height <= 5) {
        printf("Image is too small for bilateral filter (at least 5x5 size needed).\n");
        stbi_image_free(h_src);
        return 1;
    }

    // Allocation pour image filtrée (CPU) (Host)
    size_t img_size = width * height * channels * sizeof(unsigned char);
    unsigned char *h_dst = (unsigned char *)malloc(img_size);
    if (!h_dst) {
        printf("Memory allocation for filtered image failed!\n");
        stbi_image_free(h_src);
        return 1;
    }

    // Préparation GPU 

    // hipMalloc : Alloue des octets de taille de mémoire linéaire sur le périphérique et renvoie dans *devPtr un pointeur à la mémoire allouée. 
    // La mémoire allouée est convenablement alignée pour tout type de variable. 
    // La mémoire n’est pas effacée. hipMalloc() renvoie hipErrorOutOfMemory en cas d’échec.

    unsigned char *d_src = nullptr;
    unsigned char *d_dst = nullptr;
    hipMalloc(&d_src, img_size);
    hipMalloc(&d_dst, img_size);
    // permet de copier des données entre la mémoire cpu et la mémoire gpu
    hipMemcpy(d_src, h_src, img_size, hipMemcpyHostToDevice);

    // Préparation des poids spatiaux

    int filter_d = 5;
    float sigma_color = 75.0f;
    float sigma_space = 75.0f;
    int radius = filter_d / 2;

    // on va calucler les poids et on les calcule sur le CPU (car c’est simple), puis on les envoie au GPU
    float *h_spatial_weights = (float *)malloc(filter_d * filter_d * sizeof(float));
    if (!h_spatial_weights) {
        printf("Memory allocation for spatial weights failed!\n");
        stbi_image_free(h_src);
        free(h_dst);
        hipFree(d_src);
        hipFree(d_dst);
        return 1;
    }

    for (int i = 0; i < filter_d; i++) {
        for (int j = 0; j < filter_d; j++) {
            int x = i - radius;
            int y = j - radius;
            h_spatial_weights[i * filter_d + j] = expf(-(x * x + y * y) / (2.0f * sigma_space * sigma_space));
        }
    }

    float *d_spatial_weights = nullptr;
    hipMalloc(&d_spatial_weights, filter_d * filter_d * sizeof(float));
    hipMemcpy(d_spatial_weights, h_spatial_weights, filter_d * filter_d * sizeof(float), hipMemcpyHostToDevice);

    // Lancement du kernel 
    // On divise l’image en blocs de 16x16 threads et puis on appelle le kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + 15) / 16, (height + 15) / 16);

    bilateral_filter_kernel<<<gridSize, blockSize>>>(
        d_src, d_dst, width, height, channels,
        d_spatial_weights, filter_d, sigma_color
    );
    hipDeviceSynchronize();

    // Récupération du résultat

    hipMemcpy(h_dst, d_dst, img_size, hipMemcpyDeviceToHost);

    // Sauvegarde

    if (!stbi_write_png(argv[2], width, height, channels, h_dst, width * channels)) {
        printf("Error saving the image!\n");
        free(h_dst);
        stbi_image_free(h_src);
        hipFree(d_src);
        hipFree(d_dst);
        hipFree(d_spatial_weights);
        free(h_spatial_weights);
        return 1;
    }

    // Nettoyage

    // hipFree : Libère l’espace mémoire indiqué par devPtr, qui doit avoir été retourné par un appel précédent à hipMalloc() ou hipMallocPitch().
    //Sinon, ou si hipFree(devPtr) a déjà été appelé auparavant, une erreur est renvoyée. 
    //Si devPtr est 0, aucune opération n’est effectuée. hipFree() renvoie hipErrorInvalidDevicePointer en cas d’échec. (la doc nvidia)

    stbi_image_free(h_src);
    free(h_dst);
    free(h_spatial_weights);
    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_spatial_weights);

    printf("Bilateral filtering (CUDA) complete. Output saved as %s\n", argv[2]);
    return 0;
}

